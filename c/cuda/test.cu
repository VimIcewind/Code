
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
	int device_id = 0;
	hipDeviceProp_t prop;

	hipGetDeviceProperties(&prop, device_id);

	printf("Device name: %s\n", prop.name);
	printf("Compute capability: %d.%d\n", prop.major, prop.minor);

	return 0;
}
